//make sure to run as nvcc cublas.cu -lcublas


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <hipblas.h>

#define TILE_WIDTH 2

int main(){

	int width = 2 * TILE_WIDTH;
	float *x, *y, *z;
	float *hx, *hy, *hz;

	hx = (float *)malloc(sizeof(float)*width*width);
	hy = (float *)malloc(sizeof(float)*width*width);
	hz = (float *)malloc(sizeof(float)*width*width);

	hipMallocManaged(&x, sizeof(float)*width*width);
	hipMallocManaged(&y, sizeof(float)*width*width);
	hipMallocManaged(&z, sizeof(float)*width*width);

	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      hx[i * width + j] = 1.0; // x[i][j]
	      hy[i * width + j] = 1.0;
	    }
	  }

	for (int i=0;i<width;i++){
		for (int j=0;j<width;j++){
			printf("%f ", hx[i*width+j]);
			
		}
		printf("\n");
	}
	printf("\n");

	hipMemcpy(x, hx, sizeof(float)*width*width, hipMemcpyHostToDevice);
	hipMemcpy(y, hy, sizeof(float)*width*width, hipMemcpyHostToDevice);

	// Setup the execution configuration
	// TILE_WIDTH is a #define constant
	
	//dim3 dimGrid(ceil((1.0*width)/TILE_WIDTH),
	//  ceil((1.0*width)/TILE_WIDTH), 1);
	//dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	hipDeviceSynchronize();
	 // Launch the device computation threads!

	hipblasHandle_t handle;
	hipblasCreate(&handle); 

	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, width, 
	width, width, alpha, x, width, y, width, beta, z, width);

	hipblasDestroy(handle);
  	
  	printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(hz, z, sizeof(float)*width*width, hipMemcpyDeviceToHost);
	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      if (hz[i * width + j] != width) {
	        printf("Error at z[%d][%d]: %f\n", i, j,
	         hz[i * width + j]);
	      }
	    }
	  }


	hipFree(x);
	hipFree(y);
	hipFree(z);
	free(hx);
	free(hy);
	free(hz);

	return 0;
}
