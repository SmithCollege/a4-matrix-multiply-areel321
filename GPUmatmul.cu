
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 50

double get_clock() {
        struct timeval tv; int ok;
        ok = gettimeofday(&tv, (void *) 0);
        if (ok<0){
                printf("gettimeofday error");
        }
        return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}


__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width)
{
 //calculate row index of the d_P element and d_M
 int Row = blockIdx.y*blockDim.y+threadIdx.y;
 // Calculate the column idenx of d_P and d_N
 int Col = blockIdx.x*blockDim.x+threadIdx.x;
 
 if ((Row < Width) && (Col < Width)) {
	 float Pvalue = 0.0;
	 // each thread computes one element of the block sub-matrix
	 for (int k = 0; k < Width; ++k){
	  Pvalue += d_M[Row*Width+k] * d_N[k*Width+Col];
	 }
	 d_P[Row * Width + Col] = Pvalue;
  }
}


int main(){

	int width = 2 * TILE_WIDTH;
	float *x, *y, *z;
	float *hx, *hy, *hz;

	double *times = (double *)malloc(sizeof(double)*width);
	 //calibrate clock
        double t0 = get_clock();
        for (int i=0; i<width; i++){
            times[i] = get_clock();
        }
        double t1 = get_clock();
        printf("time per call: %f nx\n", (1000000000.0 * (t1-t0\
)/width));


	hx = (float *)malloc(sizeof(float)*width*width);
	hy = (float *)malloc(sizeof(float)*width*width);
	hz = (float *)malloc(sizeof(float)*width*width);

	hipMallocManaged(&x, sizeof(float)*width*width);
	hipMallocManaged(&y, sizeof(float)*width*width);
	hipMallocManaged(&z, sizeof(float)*width*width);

	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      hx[i * width + j] = 1.0; // x[i][j]
	      hy[i * width + j] = 1.0;
	    }
	  }

/*	for (int i=0;i<width;i++){
		for (int j=0;j<width;j++){
			printf("%f ", hx[i*width+j]);
			
		}
		printf("\n");
	}*/
	printf("\n");

	hipMemcpy(x, hx, sizeof(float)*width*width, hipMemcpyHostToDevice);
	hipMemcpy(y, hy, sizeof(float)*width*width, hipMemcpyHostToDevice);

	// Setup the execution configuration
	// TILE_WIDTH is a #define constant
	
	dim3 dimGrid(ceil((1.0*width)/TILE_WIDTH),
	  ceil((1.0*width)/TILE_WIDTH), 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	hipDeviceSynchronize();

	double start_time = get_clock();

	 // Launch the device computation threads!
  	MatrixMulKernel<<<dimGrid, dimBlock>>>(x,y, z, width);

	double end_time = get_clock();
	printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(hz, z, sizeof(float)*width*width, hipMemcpyDeviceToHost);

	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      if (hz[i * width + j] != width) {
	        printf("Error at z[%d][%d]: %f\n", i, j,
	         hz[i * width + j]);
	      }
	    }
	  }

	  //print clock times
        printf("start: %f, end: %f\n", start_time, end_time);


	hipFree(x);
	hipFree(y);
	hipFree(z);
	free(hx);
	free(hy);
	free(hz);
	free(times);

	return 0;
}
