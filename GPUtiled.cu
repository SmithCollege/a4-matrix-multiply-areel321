
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 50


double get_clock() {
        struct timeval tv; int ok;
        ok = gettimeofday(&tv, (void *) 0);
        if (ok<0){
                printf("gettimeofday error\n");
        }
        return (tv.tv_sec*1.0+tv.tv_usec*1.0E-6);
}

 __global__ void MatrixMulKernel(float* M, float* N, float* P, int Width)
{
 __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
 __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];
 int bx = blockIdx.x; int by = blockIdx.y;
 int tx = threadIdx.x; int ty = threadIdx.y;
 // Identify the row and column of the P element to work on
 int Row = by * TILE_WIDTH + ty;
 int Col = bx * TILE_WIDTH + tx;
 float Pvalue = 0;
 // Loop over the M and N tiles required to compute the P element
 // The code assumes that the Width is a multiple of TILE_WIDTH!
 for (int m = 0; m < Width/TILE_WIDTH; ++m) {
 // Collaborative loading of M and N tiles into shared memory
 subTileM[ty][tx] = M[Row*Width + m*TILE_WIDTH+tx];
 subTileN[ty][tx] = N[(m*TILE_WIDTH+ty)*Width+Col];
 __syncthreads();
 for (int k = 0; k < TILE_WIDTH; ++k) {
 Pvalue += subTileM[ty][k] * subTileN[k][tx];
 }
 __syncthreads();
 }
 P[Row*Width+Col] = Pvalue;
}


int main(){

	int width = 2 * TILE_WIDTH;
	float *x, *y, *z;
	float *hx, *hy, *hz;

	double *times = (double *)malloc(sizeof(double) * width);


                //calibrate the clock
        double t0 = get_clock();
        for (int i=0; i<width; i++){
                times[i] = get_clock();
        }
        double t1 = get_clock();
        printf("time per call: %f nx\n", (1000000000.0 * (t1-t0\
)/width));


	hx = (float *)malloc(sizeof(float)*width*width);
	hy = (float *)malloc(sizeof(float)*width*width);
	hz = (float *)malloc(sizeof(float)*width*width);

	hipMallocManaged(&x, sizeof(float)*width*width);
	hipMallocManaged(&y, sizeof(float)*width*width);
	hipMallocManaged(&z, sizeof(float)*width*width);

	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      hx[i * width + j] = 1.0; // x[i][j]
	      hy[i * width + j] = 1.0;
	    }
	  }

/*	for (int i=0;i<width;i++){
		for (int j=0;j<width;j++){
			printf("%f ", hx[i*width+j]);
			
		}
		printf("\n");
	}*/
	printf("\n");

	hipMemcpy(x, hx, sizeof(float)*width*width, hipMemcpyHostToDevice);
	hipMemcpy(y, hy, sizeof(float)*width*width, hipMemcpyHostToDevice);

	// Setup the execution configuration
	// TILE_WIDTH is a #define constant
	
	dim3 dimGrid(ceil((1.0*width)/TILE_WIDTH),
	  ceil((1.0*width)/TILE_WIDTH), 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	hipDeviceSynchronize();
double start = get_clock();

// Launch the device computation threads!
  	MatrixMulKernel<<<dimGrid, dimBlock>>>(x,y, z, width);
double end = get_clock();

printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(hz, z, sizeof(float)*width*width, hipMemcpyDeviceToHost);
	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      if (hz[i * width + j] != width) {
	        printf("Error at z[%d][%d]: %f\n", i, j,
	         hz[i * width + j]);
	      }
	    }
	  }

//print clock times
        printf("start: %f, end: %f\n", start, end);

	hipFree(x);
	hipFree(y);
	hipFree(z);
	free(hx);
	free(hy);
	free(hz);
free(times);

	return 0;
}
