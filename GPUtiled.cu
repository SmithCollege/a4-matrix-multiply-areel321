
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#define TILE_WIDTH 2


 __global__ void MatrixMulKernel(float* M, float* N, float* P, int Width)
{
 __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
 __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];
 int bx = blockIdx.x; int by = blockIdx.y;
 int tx = threadIdx.x; int ty = threadIdx.y;
 // Identify the row and column of the P element to work on
 int Row = by * TILE_WIDTH + ty;
 int Col = bx * TILE_WIDTH + tx;
 float Pvalue = 0;
 // Loop over the M and N tiles required to compute the P element
 // The code assumes that the Width is a multiple of TILE_WIDTH!
 for (int m = 0; m < Width/TILE_WIDTH; ++m) {
 // Collaborative loading of M and N tiles into shared memory
 subTileM[ty][tx] = M[Row*Width + m*TILE_WIDTH+tx];
 subTileN[ty][tx] = N[(m*TILE_WIDTH+ty)*Width+Col];
 __syncthreads();
 for (int k = 0; k < TILE_WIDTH; ++k) {
 Pvalue += subTileM[ty][k] * subTileN[k][tx];
 }
 __syncthreads();
 }
 P[Row*Width+Col] = Pvalue;
}


int main(){

	int width = 2 * TILE_WIDTH;
	float *x, *y, *z;
	float *hx, *hy, *hz;

	hx = (float *)malloc(sizeof(float)*width*width);
	hy = (float *)malloc(sizeof(float)*width*width);
	hz = (float *)malloc(sizeof(float)*width*width);

	hipMallocManaged(&x, sizeof(float)*width*width);
	hipMallocManaged(&y, sizeof(float)*width*width);
	hipMallocManaged(&z, sizeof(float)*width*width);

	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      hx[i * width + j] = 1.0; // x[i][j]
	      hy[i * width + j] = 1.0;
	    }
	  }

	for (int i=0;i<width;i++){
		for (int j=0;j<width;j++){
			printf("%f ", hx[i*width+j]);
			
		}
		printf("\n");
	}
	printf("\n");

	hipMemcpy(x, hx, sizeof(float)*width*width, hipMemcpyHostToDevice);
	hipMemcpy(y, hy, sizeof(float)*width*width, hipMemcpyHostToDevice);

	// Setup the execution configuration
	// TILE_WIDTH is a #define constant
	
	dim3 dimGrid(ceil((1.0*width)/TILE_WIDTH),
	  ceil((1.0*width)/TILE_WIDTH), 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	hipDeviceSynchronize();
	 // Launch the device computation threads!
  	MatrixMulKernel<<<dimGrid, dimBlock>>>(x,y, z, width);
  	printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(hz, z, sizeof(float)*width*width, hipMemcpyDeviceToHost);
	for (int i = 0; i < width; i++) {
	    for (int j = 0; j < width; j++) {
	      if (hz[i * width + j] != width) {
	        printf("Error at z[%d][%d]: %f\n", i, j,
	         hz[i * width + j]);
	      }
	    }
	  }


	hipFree(x);
	hipFree(y);
	hipFree(z);
	free(hx);
	free(hy);
	free(hz);

	return 0;
}
